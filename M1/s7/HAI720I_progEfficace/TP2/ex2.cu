#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>

using namespace std;

#define cudaCheckError() {                                                                       \
        hipError_t e=hipGetLastError();                                                        \
        if(e!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
            exit(EXIT_FAILURE);                                                                  \
        }                                                                                        \
    }

__global__ void vectorAddition(float* C, const float* A, const float* B, size_t n) {
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if(i < n){
    C[i]=A[i]+B[i];
  }
}

void vectorFloatSum(float* C, float* A, float* B, int n){
  for(int i = 0; i < n; i++){
    C[i] = A[i] + B[i];
  }
}


void prepareCallVecFloatSum(int nbVal){

  float* A = (float*) malloc(nbVal*sizeof(float));
  float* B = (float*) malloc(nbVal*sizeof(float));
  float* C = (float*) malloc(nbVal*sizeof(float));

  for(int i = 0; i < nbVal; i++){
    A[i] = (float) (rand() % 65535);
    B[i] = (float) (rand() % 65535);
  }

  vectorFloatSum(C, A, B, nbVal);

  for (int i = 0; i < nbVal; i++)
  {
    cout << C[i] << endl;;
  }

  free(A);
  free(B);
  free(C);
}


void vectorFloatSumCUDA(float *h_A, float *h_B, float *h_C, int n){
  int threads=128;
  int blocks=(n+threads-1)/threads;
  int size = n * sizeof(float);
  float *d_A, *d_B, *d_C;

  hipMallocManaged(&d_A,size);
  hipMallocManaged(&d_B,size);
  hipMallocManaged(&d_C,size);

  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  vectorAddition<<<blocks,threads>>>(d_C, d_A, d_B, n);
  hipDeviceSynchronize();

  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  cudaCheckError();
  
}

void CUDAprepareCallVecFloatSum(){
  int N=1024;
  float* A = (float*) malloc(1024*sizeof(float));
  float* B = (float*) malloc(1024*sizeof(float));
  float* C = (float*) malloc(1024*sizeof(float));

  for(int i = 0; i < N; i++){
    A[i] = (float)i;
    B[i] = (float)(N-i);
  }
  vectorFloatSumCUDA(A,B,C,N);
  for(int i=0;i<10;i++){
    printf("%f\n",C[i]);
  }

  free(A);
  free(B);
  free(C);
}

__global__ void matrixMulKernel(float* M, const float* N, const float* P, int width) {
  int row = blockIdx.y*blockDim.y+threadIdx.y;
  int col = blockIdx.x*blockDim.x+threadIdx.x;
  if((row < width) && (col < width)) {
    float Pvalue = 0;
    // each thread computes one element of the block sub-matrix
    for (int k = 0; k < width; ++k)
    {
      Pvalue += M[row*width+k]*N[k*width+col];
    }
    P[row*width+col] = Pvalue;
  }
}

void matrixMulCUDA(float *h_A, float *h_B, float *h_C, int n){
  int threads=128;
  int blocks=(n+threads-1)/threads;
  int size = n * sizeof(float);
  float *d_A, *d_B, *d_C;

  hipMallocManaged(&d_A,size);
  hipMallocManaged(&d_B,size);
  hipMallocManaged(&d_C,size);

  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  matrixMulKernel<<<blocks,threads>>>(d_C, d_A, d_B, n);
  hipDeviceSynchronize();

  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  cudaCheckError();
  
}


int main() {
  prepareCallVecFloatSum(1024);
  CUDAprepareCallVecFloatSum();
  return 0;
}
